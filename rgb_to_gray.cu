#include "hip/hip_runtime.h"
#include "header.h"

static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number){
	if(err!=hipSuccess){
		fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",msg,file_name,line_number,hipGetErrorString(err));
		std::cin.get();
		exit(EXIT_FAILURE);
	}
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)

extern "C"
void convert_to_gray_cpu(unsigned char* input,int rows, int cols, int color_step, unsigned char* output, int gray_step);

extern "C"
void convert_to_gray_gpu(const cv::Mat& input, cv::Mat& output);

__global__ void rgb_to_gray_kernel(unsigned char* input, unsigned char* output, unsigned int pixel_cnt){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if( tid < pixel_cnt ){
		int color_idx = tid*BYTES_PER_PIXEL;
		int gray_idx = tid;
		unsigned char blue	= input[color_idx];
		unsigned char green	= input[color_idx + 1];
		unsigned char red	= input[color_idx + 2];
		float gray = red * 0.3f + green * 0.59f + blue * 0.11f;
		output[gray_idx] = static_cast<unsigned char>(gray);
	}
}

void convert_to_gray_cpu(unsigned char* input,int rows, int cols, int color_step, unsigned char* output, int gray_step){

	float milliseconds = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	unsigned char blue;
	unsigned char green;
	unsigned char red;
	float gray;

	int color_idx = 0;
	int gray_idx = 0;
	for( int i=0; i<rows; i++ ){
		for(int j=0; j<cols; j++){
			color_idx = i*color_step + (3*j);
			gray_idx = i*gray_step + j;
			blue	= input[color_idx];
			green	= input[color_idx + 1];
			red		= input[color_idx + 2];
			gray = red * 0.3f + green * 0.59f + blue * 0.11f;
			output[gray_idx] = static_cast<unsigned char>(gray);
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout<<"CPU time: "<<milliseconds<<" ms"<<std::endl;
}

void convert_to_gray_gpu(const cv::Mat& input, cv::Mat& output){
	const int colorBytes = input.step * input.rows;
	const int grayBytes = output.step * output.rows;
	unsigned char *d_input, *d_output;

	float milliseconds = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// GPU processing......
	hipEventRecord(start);

	//Allocate device memory
	SAFE_CALL(hipMalloc(&d_input,colorBytes),"CUDA Malloc Failed");
	SAFE_CALL(hipMalloc(&d_output,grayBytes),"CUDA Malloc Failed");

	//Copy data from OpenCV input image to device memory
	SAFE_CALL(hipMemcpy(d_input,input.ptr(),colorBytes,hipMemcpyHostToDevice),"CUDA Memcpy Host To Device Failed");

	//Specify a reasonable block size
	dim3 block(BLOCK_SIZE);

	//Calculate grid size to cover the whole image
	dim3 grid((input.cols * input.rows + BLOCK_SIZE -1 )/BLOCK_SIZE);

	rgb_to_gray_kernel<<<grid, block>>>(d_input, d_output, input.cols*input.rows);

	//Synchronize to check for any kernel launch errors
	SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed");
	//Copy back data from destination device meory to OpenCV output image
	SAFE_CALL(hipMemcpy(output.ptr(),d_output,grayBytes,hipMemcpyDeviceToHost),"CUDA Memcpy Device To Host Failed");

	//Free the device memory
	SAFE_CALL(hipFree(d_input),"CUDA Free Failed");
	SAFE_CALL(hipFree(d_output),"CUDA Free Failed");

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout<<"GPU time: "<<milliseconds<<" ms"<<std::endl;
}
